#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <stdio.h>
#include <iostream>
#define THREADS 1024

const double t1 = 500000; // 50000
const int N = 500; // 10000			//	����������� ��������� �� ������� �������� �������� 
const double t_final = 600;	//	����� ������������ ��������
const double L = 0.1;		//	����� �������
const double lambda = 46;	//	��������� ������ � ��������� ����������������
const double ro = 7800;		//	��������� � �������� ����������������
const double c = 460;		//	������������ ��������� � ��������� ����������������
const double T_0 = 20;		//	��������� �����������
const double T_l = 300;		//	����������� �� ������ �=0
const double T_r = 100;		//	����������� �� ������� �=L

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void yavn(double time, double h, double* T_arr, double a, double tau, double T_l, double T_r, double t_final)
{
	int i = threadIdx.x;
	double* T_prev = T_arr;
	T_arr[0] = T_l;
	T_arr[N - 1] = T_r;
	for (double j = time; j < t_final; j += tau)
	{
		T_prev = T_arr;
		//for (int i = 1; i < N - 1; i++)
		if (i < N -1) 
			T_arr[i] = T_prev[i] + a*tau / pow(h, 2)*(T_prev[i + 1] - 2 * T_prev[i] + T_prev[i - 1]);
		
	}
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{

	auto h_step = [](double L, int N) { return L / (N - 1); };
	double h = h_step(L, N);
	double *x = (double *)malloc(40000 * sizeof(double));
	double time = 0;
	double *T_arr;
	double *res;
	double a = lambda / (ro*c);
	double tau = 0.25 * pow(h, 2) / a; /// �� ������� ������������

	hipMalloc((void **)&T_arr, sizeof(double));
	hipMemcpy(T_arr, x, sizeof(double), hipMemcpyHostToDevice);
	yavn << <1, THREADS >> > (time, h, T_arr, a, tau, T_l, T_r, t_final);
	hipMemcpy(&res, T_arr, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(T_arr);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
